#include "hip/hip_runtime.h"
#include "rc/Control.cuh"

void calcState(uint16_t *laserDat, uint8_t *states) {}

void agentUpdate(float *qtable, uint8_t *cstate, uint8_t *nstate, float *reward,
                 uint8_t *action) {}

void agentAction(float *qtable, uint8_t *cstate, uint8_t *action) {}

void agentReward(uint8_t *cstate, uint8_t *nstate, float *reward) {
    for (int i = 0; i < NUM_REGIONS; i++) {
        if (abs(nstate[i] - CTR_STATE) != abs(cstate[i] - CTR_STATE)) { //got either closer or farther to center state
            reward[i] = 1.0 + log(abs(nstate[i] - cstate[i])); //minimum of 1 reward, diminishing returns after
            if (abs(nstate[i] - CTR_STATE) > abs(cstate[i] - CTR_STATE)) { //nextstate got farther from center
                reward[i] *= -1.0;
            }
        } else { //state maintained
            reward = 0.0;
        }
    }
}

void initvals(float *qtable) {}

#include "hip/hip_runtime.h"
#include "rc/Control.cuh"

__device__ int qtableAccessor(uint8_t *state) {
  int qtableIndex = 0;
  for (int i = 0; i < NUM_REGIONS; i++)
    qtableIndex += state[i] * (NUM_STATES ^ i);
  return qtableIndex * 4;
}

__device__ int myMax(int a, int b) { return (a > b) ? a : b; }

__global__ void findState(uint16_t *laserDat, uint8_t *states) {
  // parallelized by region - tid is region num
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int sum = 0;
  int offset = 0;
  int avg, tmp;
  for (size_t i = 0; i < LIDAR_VALS / NUM_REGIONS; i++) {
    // iterate through region
    tmp = laserDat[tid * NUM_REGIONS + i];
    // if bad value, discard
    if (0 == tmp || LIDAR_MAX_V < tmp) {
      tmp = 0;
      offset++;
    }
    sum += tmp;
  }
  // avg of region is sum / numvals
  avg = sum / ((LIDAR_VALS / NUM_REGIONS) - offset);
  int stateSize = LIDAR_MAX_V / NUM_STATES;
  // assumes target is exactly half of max v
  for (int i = 0; i < NUM_STATES; i++) {
    if (avg < stateSize * i) {
      states[tid] = i + 1;
      return;
    }
  }
}

void calcState(uint16_t *laserDat, uint8_t *states) {
  findState<<<1, NUM_REGIONS>>>(laserDat, states);
}

__global__ void deviceUpdate(float *qtable, uint8_t *cstate, uint8_t *nstate,
                             float *reward, uint8_t *action) {
  short maxA = 0;
  float nMaxVal = qtable[qtableAccessor(nstate) + maxA];
  for (int i = 0; i < NUM_ACTION; i++) {
    if (qtable[qtableAccessor(nstate) + i] > nMaxVal) {
      maxA = i;
      nMaxVal = qtable[qtableAccessor(nstate) + i];
    }
  }
  // qtable CState index
  int qtCIdx = qtableAccessor(cstate) + *action;
  // qtable maxA NState index
  int qtNIdx = qtableAccessor(nstate) + maxA;

  if (0 == *reward)
    qtable[qtCIdx] += ((*reward + DISC_FACT * qtable[qtNIdx] - qtable[qtCIdx]) /
                       LEARN_RATE_DIV);
  else
    qtable[qtCIdx] += (*reward - qtable[qtCIdx]) / LEARN_RATE_DIV;
}

void agentUpdate(float *qtable, uint8_t *cstate, uint8_t *nstate, float *reward,
                 uint8_t *action) {
  deviceUpdate<<<1, 1>>>(qtable, cstate, nstate, reward, action);
}

__global__ void deviceAction(float *qtable, uint8_t *cstate, uint8_t *action) {
  int qtableIndex = qtableAccessor(cstate);
  float currMax = qtable[qtableIndex];
  int newMax = 0;
  uint8_t currGuess = 0;
  for (int i = 0; i < 4; i++) {
    newMax = myMax(currMax, qtable[qtableIndex + i]);
    if (newMax > currMax) {
      currMax = newMax;
      currGuess = (uint8_t)i;
    }
  }
  *action = currGuess;
}

void agentAction(float *qtable, uint8_t *cstate, uint8_t *action) {
  deviceAction<<<1, 1>>>(qtable, cstate, action);
}

__global__ void getReward(uint8_t *cstate, uint8_t *nstate, float *reward) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (abs(nstate[tid] - CTR_STATE) != abs(cstate[tid] - CTR_STATE)) {
    // got either closer or farther to center state
    reward[tid] = 1.0 + abs(nstate[tid] - cstate[tid]) / NUM_STATES;
    // minimum of 1 reward, diminishing returns after
    if (abs(nstate[tid] - CTR_STATE) > abs(cstate[tid] - CTR_STATE)) {
      // nextstate got farther from center
      reward[tid] *= -1.0;
    }
  } else { // state maintained
    reward[tid] = 0.0;
  }
}

// avg out reward, store into first value of reward
__global__ void avgReward(float *reward) {
  float sum = 0;
  for (int i = 0; i < NUM_REGIONS; i++) {
    sum += reward[i];
  }
  reward[0] = sum / NUM_REGIONS;
}

void agentReward(uint8_t *cstate, uint8_t *nstate, float *reward) {
  // TODO make sure this is not bad;
  getReward<<<1, NUM_REGIONS>>>(cstate, nstate, reward);
  avgReward<<<1, 1>>>(reward);
}

__global__ void initQtable(float *qtable) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  qtable[tid] = 0;
}
void initvals(float *qtable) {
  size_t qtabSize = (NUM_STATES ^ NUM_REGIONS) * NUM_ACTION;
  initQtable<<<qtabSize / 64, 64>>>(qtable);
}

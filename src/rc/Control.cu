#include "hip/hip_runtime.h"
#include "rc/Control.cuh"
#include <cstdlib>

__global__ void findState(uint16_t *laserDat, uint8_t *states) {
  // parallelized by region - tid is region num
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int sum = 0;
  int offset = 0;
  int avg, tmp;
  for (size_t i = 0; i < LIDAR_VALS / NUM_REGIONS; i++) {
    // iterate through region
    tmp = laserDat[tid * NUM_REGIONS + i];
    // if bad value, discard
    if (0 == tmp || LIDAR_MAX_V < tmp) {
      tmp = 0;
      offset++;
    }
    sum += tmp;
  }
  // avg of region is sum / numvals
  avg = sum / ((LIDAR_VALS / NUM_REGIONS) - offset);
  int stateSize = LIDAR_MAX_V / NUM_STATES;
  // assumes target is exactly half of max v
  for (int i = 0; i < NUM_STATES; i++) {
    if (avg < stateSize * i) {
      states[tid] = i + 1;
      return;
    }
  }
}

void calcState(uint16_t *laserDat, uint8_t *states) {
  findState<<<1, NUM_REGIONS>>>(laserDat, states);
}

void agentUpdate(float *qtable, uint8_t *cstate, uint8_t *nstate, float *reward,
                 uint8_t *action) {}

void agentAction(float *qtable, uint8_t *cstate, uint8_t *action) {}

void agentReward(uint8_t *cstate, uint8_t *nstate, float *reward) {
    for (int i = 0; i < NUM_REGIONS; i++) {
        if (abs(nstate[i] - CTR_STATE) != abs(cstate[i] - CTR_STATE)) { //got either closer or farther to center state
            reward[i] = 1.0 + log(abs(nstate[i] - cstate[i])); //minimum of 1 reward, diminishing returns after
            if (abs(nstate[i] - CTR_STATE) > abs(cstate[i] - CTR_STATE)) { //nextstate got farther from center
                reward[i] *= -1.0;
            }
        } else { //state maintained
            reward = 0.0;
        }
    }
}

__global__ void initQtable(float *qtable) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  qtable[tid] = 0;
}
void initvals(float *qtable) {
  size_t qtabSize = (NUM_STATES ^ NUM_REGIONS) * NUM_ACTION;
  initQtable<<<qtabSize / 64, 64>>>();
}

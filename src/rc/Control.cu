#include "hip/hip_runtime.h"
#include "rc/Control.cuh"
#include <cstdio>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// region number of values
#define REG_NV (LIDAR_VALS / LIDAR_DIV) / NUM_REGIONS

__device__ int qtableAccessor(uint8_t *state) {
  int qtableIndex = 0;
  for (int i = 0; i < NUM_REGIONS; i++)
    qtableIndex += state[i] * pow(NUM_STATES, i);
  return qtableIndex * 4;
}

__global__ void init_randstate(hiprandState *state) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(clock() + tid, tid, 0, &state[tid]);
}

void initrand(hiprandState *state) { init_randstate<<<1, 1>>>(state); }

__device__ float myMax(float a, float b) { return (a > b) ? a : b; }

__global__ void findState(uint16_t *laserDat, uint8_t *states) {
  // parallelized by region - tid is region num
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int sum = 0;
  int offset = 0;
  int avg, tmp;
  for (size_t i = 0; i < REG_NV; i++) {
    // iterate through region
    tmp = laserDat[tid * REG_NV + i];
    // if bad value, discard
    if (0 == tmp) {
      tmp = 0;
      offset++;
    } else if (LIDAR_MAX_V < tmp) {
      tmp = LIDAR_MAX_V;
    }
    sum += tmp;
  }
  // avg of region is sum / numvals
  avg = sum / (REG_NV - offset);
  int stateSize = LIDAR_MAX_V / NUM_STATES;
  // assumes target is exactly half of max v
  for (int i = 1; i <= NUM_STATES; i++) {
    if (avg <= stateSize * i) {
      states[tid] = i - 1;
      //printf(
      //    "current state for region %d is %d\nstateSize is %d and avg is %d\n",
      //    tid, i, stateSize * i, avg);
      return;
    }
  }
}

void calcState(uint16_t *laserDat, uint8_t *states) {
  findState<<<1, NUM_REGIONS>>>(laserDat, states);
}

__global__ void deviceUpdate(float *qtable, uint8_t *cstate, uint8_t *nstate,
                             float *reward, uint8_t *action) {
  short maxA = 0;
  float nMaxVal = qtable[qtableAccessor(nstate) + maxA];
  for (int i = 0; i < NUM_ACTION; i++) {
    if (qtable[qtableAccessor(nstate) + i] > nMaxVal) {
      maxA = i;
      nMaxVal = qtable[qtableAccessor(nstate) + i];
    }
  }
  // qtable CState index
  int qtCIdx = qtableAccessor(cstate) + *action;
  // qtable maxA NState index
  int qtNIdx = qtableAccessor(nstate) + maxA;

  if (0 == *reward)
    qtable[qtCIdx] += ((*reward + DISC_FACT * qtable[qtNIdx] - qtable[qtCIdx]) *
                       LEARN_RATE_DIV);
  else
    qtable[qtCIdx] += (*reward - qtable[qtCIdx]) * LEARN_RATE_DIV;
}

void agentUpdate(float *qtable, uint8_t *cstate, uint8_t *nstate, float *reward,
                 uint8_t *action) {
  deviceUpdate<<<1, 1>>>(qtable, cstate, nstate, reward, action);
}

__global__ void deviceAction(float *qtable, uint8_t *cstate, uint8_t *action,
                             hiprandState *aState) {

  // TODO: parallelize
  float rand = hiprand_uniform(aState);
  if (rand < EPSILON) {
    rand = hiprand_uniform(aState);
    *action = (short)(rand * 3.99);
    return;
  }

  int qtableIndex = qtableAccessor(cstate);
  float currMax = qtable[qtableIndex];
  float newMax = currMax;
  uint8_t currGuess = 0;
  for (int i = 0; i < 4; i++) {
    newMax = myMax(currMax, qtable[qtableIndex + i]);
    if (newMax > currMax) {
      currMax = newMax;
      currGuess = (uint8_t)i;
    }
  }
  //printf("CurrAction: %d\ncurrMax: %f\n", *action, currMax);
  *action = currGuess;
  if (currGuess == 0 && currMax == 0) {
    *action = (short)(rand * 3.99);
    //printf("rolling random%d\n", (int) *action);
  }
}

void agentAction(float *qtable, uint8_t *cstate, uint8_t *action,
                 hiprandState *astate) {
  deviceAction<<<1, 1>>>(qtable, cstate, action, astate);
}

__global__ void getReward(uint8_t *cstate, uint8_t *nstate, float *reward) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (abs(nstate[tid] - CTR_STATE) != abs(cstate[tid] - CTR_STATE)) {
    // got either closer or farther to center state
    reward[tid] = 1.0 + abs(nstate[tid] - cstate[tid]) / NUM_STATES;
    // minimum of 1 reward, diminishing returns after
    if (abs(nstate[tid] - CTR_STATE) > abs(cstate[tid] - CTR_STATE)) {
      // nextstate got farther from center
      reward[tid] *= -1.0;
    }
  } else { // state maintained
    reward[tid] = 0.0;
  }
}

// determine reward with policy, store into first value of reward
__global__ void weightReward(float *reward) {
  reward[0] = reward[NUM_REGIONS / 2];
}

void agentReward(uint8_t *cstate, uint8_t *nstate, float *reward) {
  // TODO make sure this is not bad;
  getReward<<<1, NUM_REGIONS>>>(cstate, nstate, reward);
  weightReward<<<1, 1>>>(reward);
}

// returns carry flag
// NUM_STATES - 1 is max possible value of state
// NUM_REGIONS is length of state array
// if at max value, does not iterate
__device__ bool iterateState(uint8_t *state, uint8_t targetIndex) {
  if (state[targetIndex] < NUM_STATES - 1) {
    // iterate index if index is not max value
    state[targetIndex]++;
    return false;
  } else if (state[targetIndex] == NUM_STATES - 1) {
    // else if it is, reset current index to zero, iterate next index
    if (targetIndex < NUM_REGIONS - 1) {
      state[targetIndex] = 0;
      return iterateState(state, targetIndex + 1);
    } else {
      // return false if at maximum state val
      return true;
    }
  }
  return true;
}

__global__ void initQtable(float *qtable) {
  uint8_t allstates[NUM_REGIONS];
  do {
    for (int i = 0; i < NUM_ACTION; i++) {
      qtable[qtableAccessor(allstates) + i] = 0;
    }
    if (allstates[NUM_REGIONS / 2] > CTR_STATE) {
      qtable[qtableAccessor(allstates) + ROBOT_THUP] = 1;
      qtable[qtableAccessor(allstates) + ROBOT_THDN] = -1;
    } else if (allstates[NUM_REGIONS / 2] < CTR_STATE) {
      qtable[qtableAccessor(allstates) + ROBOT_THUP] = -1;
      qtable[qtableAccessor(allstates) + ROBOT_THDN] = 1;
    }
    if (allstates[CTR_STATE - 1] < allstates[CTR_STATE]) {
      qtable[qtableAccessor(allstates) + ROBOT_STRL] = BIAS_MULT * 1;
      qtable[qtableAccessor(allstates) + ROBOT_STRR] = BIAS_MULT * -1;
    } else if (allstates[CTR_STATE + 1] > allstates[CTR_STATE]) {
      qtable[qtableAccessor(allstates) + ROBOT_STRL] = BIAS_MULT * -1;
      qtable[qtableAccessor(allstates) + ROBOT_STRR] = BIAS_MULT * 1;
    }
  } while (iterateState(allstates, 0) != true);
}

__global__ void printQTable(float *qtable) {
  for (int i = 0; i < pow(NUM_STATES, NUM_REGIONS) * NUM_ACTION; i++) {
    printf("%d: %f\n", i, qtable[i]);
  }
}
void initvals(float *qtable) {
  initQtable<<<1, 1>>>(qtable);
  hipDeviceSynchronize();
  printQTable<<<1, 1>>>(qtable);
}
